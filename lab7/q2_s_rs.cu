// Write a CUDA program that reads a string S and produces the string RS as follows:
// Input string $: PCAP Output string RS: PCAPPCAPCP
// Note: Each work item copies required number of characters from S in RS

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>


__global__ void string_process(char *s, char *rs, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        int offset = (i * (2*n - i + 1)) / 2;
        for(int k = 0; k < n-i; k++) {
            rs[offset + k] = s[k];
        }
    }
}

int main() {
    char s[100], rs[100*101/2];
    printf("Enter a string: ");
    fgets(s, 100, stdin);
    s[strcspn(s, "\n")] = 0;

    int n = strlen(s);
    int total_size = (n * (n + 1)) / 2;
    char *d_s, *d_rs;

    hipMalloc((void**)&d_s, n * sizeof(char));
    hipMalloc((void**)&d_rs, total_size * sizeof(char));

    hipMemcpy(d_s, s, n * sizeof(char), hipMemcpyHostToDevice);

    string_process<<<1, n>>>(d_s, d_rs, n);

    hipMemcpy(rs, d_rs, total_size * sizeof(char), hipMemcpyDeviceToHost);
    
    rs[total_size] = '\0';
    printf("Output string: %s\n", rs);

    hipFree(d_s);
    hipFree(d_rs);

    return 0;
}

// student@dbl-35:~/Documents/220962448/lab7$ ./out/q2
// Enter a string: PCAP
// Output string: PCAPPCAPCP
