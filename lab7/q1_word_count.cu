#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void count_words(char *sentence, char *target, int *start_indices, int num_words, int *wordCount, int targetLen, int sentenceLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_words) {
        int startIdx = start_indices[idx];
        int wordLen = (idx + 1 < num_words) ? (start_indices[idx + 1] - startIdx - 1) : (sentenceLen - startIdx);

        if (wordLen == targetLen) {
            bool match = true;
            for (int i = 0; i < wordLen && match; i++) {
                if (sentence[startIdx + i] != target[i])
                    match = false;
            }
            if (match)
                atomicAdd(wordCount, 1);
        }
    }
}

int main() {
    char sentence[100], target[100];
    printf("Enter a sentence: ");
    fgets(sentence, 100, stdin);
    sentence[strcspn(sentence, "\n")] = 0;

    printf("Enter the target: ");
    fgets(target, 100, stdin);
    target[strcspn(target, "\n")] = 0;

    int n = strlen(sentence);
    int targetLen = strlen(target);
    int wordCount = 0;

    int max_words = 50;
    int start_indices[max_words];
    int num_words = 0;
    
    int prev = 0;
    for (int i = 0; i <= n && num_words < max_words; i++) {
        if (sentence[i] == ' ' || sentence[i] == '\0') {
            if (i > prev) {
                start_indices[num_words] = prev;
                num_words++;
            }
            prev = i + 1;
        }
    }

    char *d_sentence, *d_target;
    int *d_start_indices, *d_wordCount;
    
    hipMalloc((void**)&d_sentence, 100 * sizeof(char));
    hipMalloc((void**)&d_target, 100 * sizeof(char));
    hipMalloc((void**)&d_start_indices, max_words * sizeof(int));
    hipMalloc((void**)&d_wordCount, sizeof(int));
    
    hipMemcpy(d_sentence, sentence, 100 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_target, target, 100 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_start_indices, start_indices, max_words * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_wordCount, &wordCount, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (num_words + threadsPerBlock - 1) / threadsPerBlock;
    
    count_words<<<blocks, threadsPerBlock>>>(d_sentence, d_target, d_start_indices, num_words, d_wordCount, targetLen, n);
    
    hipDeviceSynchronize();
    hipMemcpy(&wordCount, d_wordCount, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Number of occurrences: %d\n", wordCount);

    hipFree(d_sentence);
    hipFree(d_target);
    hipFree(d_start_indices);
    hipFree(d_wordCount);
    
    return 0;
}

// student@dbl-35:~/Documents/220962448/lab7$ ./out/q1
// Enter a sentence: this is a test is a is
// Enter the target: is
// Number of occurrences: 3