
#include <hip/hip_runtime.h>
// // Write a program in CUDA to count the number of times a given word is repeated in a sentence.
// // (Use Atomic function)

// #include <stdio.h>
// #include <cuda_runtime.h>
// #include <device_launch_parameters.h>

// __global__ void count_words(char *sentence, char* target, int startIdx, int wordLen, int *wordCount, int targetLen) {
//     if (wordLen == targetLen) {
//         for(int i = startIdx; i < startIdx+wordLen; i++) {
//             if(sentence[i] != target[i-startIdx]) {
//                 return;
//             }
//         }
//         atomicAdd(wordCount, 1);
//     }
// }

// int main() {
//     char sentence[100], target[100];
//     printf("Enter a sentence: ");
//     fgets(sentence, 100, stdin);
//     sentence[strcspn(sentence, "\n")] = 0;

//     printf("Enter the target: ");
//     fgets(target, 100, stdin);
//     target[strcspn(target, "\n")] = 0;

//     int n = strlen(sentence);
//     int targetLen = strlen(target);
//     int wordCount=0;
//     dim3 blk(ceil(n/3.0), 1, 1);
//     dim3 thr(3, 1, 1);

//     char *d_sentence, *d_target;
//     int *d_wordCount;
    
//     cudaMalloc((void**)&d_sentence, 100 * sizeof(char));
//     cudaMalloc((void**)&d_target, 100 * sizeof(char));
//     cudaMalloc((void**)&d_wordCount, sizeof(int));
    
//     cudaMemcpy(d_sentence, sentence, 100 * sizeof(char), cudaMemcpyHostToDevice);
//     cudaMemcpy(d_target, target, 100 * sizeof(char), cudaMemcpyHostToDevice);
//     cudaMemcpy(d_wordCount, &wordCount, sizeof(int), cudaMemcpyHostToDevice);

//     int prev = 0;
//     for (int i = 0; i <= n; i++) {
//         if(sentence[i] == ' ' || sentence[i] == '\0') {
//             if (i > prev) {
//                 count_words<<<1,1>>>(d_sentence, d_target, prev, i - prev, d_wordCount, targetLen);
//                 cudaDeviceSynchronize();
//             }
//             prev = i + 1;
//         }
//     }
    
//     cudaMemcpy(&wordCount, d_wordCount, sizeof(int), cudaMemcpyDeviceToHost);
    
//     printf("Number of occurences: %d\n", wordCount);

//     cudaFree(d_sentence);
//     cudaFree(d_target);
//     cudaFree(d_wordCount);
// }
