#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void matmul_row(int *A, int *B, int *C, int hA, int wA, int wB) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < hA) {
        for(int col = 0; col < wB; col++) {
            int elem = 0;
            for(int k = 0; k < wA; k++) {
                elem += A[row*wA + k] * B[k*wB + col];
            }
            C[row*wB + col] = elem;
        }
    }
}

__global__ void matmul_col(int *A, int *B, int *C, int hA, int wA, int wB) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < wB) {
        for(int row = 0; row < hA; row++) {
            int elem = 0;
            for(int k = 0; k < wA; k++) {
                elem += A[row*wA + k] * B[k*wB + col];
            }
            C[row*wB + col] = elem;
        }
    }
}

__global__ void matmul_elem(int *A, int *B, int *C, int hA, int wA, int wB) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < hA && col < wB) {
        int elem = 0;
        for(int k = 0; k < wA; k++) {
            elem += A[row*wA + k] * B[k*wB + col];
        }
        C[row*wB + col] = elem;
    }
}

int main() {
    int A[5][4] = {
        {3, 7, 1, 8},
        {2, 9, 4, 5},
        {6, 0, 8, 2},
        {9, 3, 6, 1},
        {4, 5, 7, 9}
    };
    int B[4][6] = {
        {2, 8, 1, 6, 9, 3},
        {7, 4, 9, 0, 5, 8},
        {6, 2, 3, 7, 1, 4},
        {9, 5, 8, 3, 7, 6}
    };

    int hA = 5, wA = 4, wB = 6;
    int *d_A, *d_B, *d_C, C[hA][wB];

    hipMalloc((void **)&d_A, hA*wA*sizeof(int));
    hipMalloc((void **)&d_B, wA*wB*sizeof(int));
    hipMalloc((void **)&d_C, hA*wB*sizeof(int));

    hipMemcpy(d_A, A, hA*wA*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, wA*wB*sizeof(int), hipMemcpyHostToDevice);

    // a:
    // dim3 threads_a(32, 1, 1);
    // dim3 blocks_a(ceil(h / 32.0), 1, 1);
    // matmul_row<<<blocks_a, threads_a>>>(d_A, d_B, d_C, hA, wA, wB);

    // b:
    // dim3 threads_b(32, 1, 1);
    // dim3 blocks_b(ceil(w / 32.0), 1, 1);
    // matmul_col<<<blocks_b, threads_b>>>(d_A, d_B, d_C, hA, wA, wB);

    // c:
    dim3 threads_c (32, 32, 1);
    dim3 blocks_c (ceil(hA/32.0), ceil(wB/32.0), 1);
    matmul_elem<<<blocks_c, threads_c>>>(d_A, d_B, d_C, hA, wA, wB);


    hipMemcpy(C, d_C, hA*wB*sizeof(int), hipMemcpyDeviceToHost);
    printf("Result: ");
    for(int i = 0; i < hA; i++) {
        for(int j = 0; j < wB; j++) {
            printf("%d  ", C[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

// Result: 133  94  133  49  119  117  
// 136  85  135  55  102  124  
// 78  74  46  98  76  62  
// 84  101  62  99  109  81  
// 166  111  142  100  131  134 