#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void matadd_row(int *A, int *B, int *C, int h, int w) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < h) {
        for(int col = 0; col < w; col++) {
            int elem = A[row*h + col] + B[row*h + col];
            C[row*h + col] = elem;
        }

    }
}

__global__ void matadd_col(int *A, int *B, int *C, int h, int w) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < w) {
        for(int row = 0; row < h; row++) {
            int elem = A[row*h + col] + B[row*h + col];
            C[row*h + col] = elem;
        }

    }
}

__global__ void matadd_elem(int *A, int *B, int *C, int h, int w) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < h && col < w) {
        int elem = A[row*h + col] + B[row*h + col];
        C[row*h + col] = elem;
    }
}



int main() {
    int A[4][4] = {
        {5, 15, 2, 14},
        {9, 13, 13, 5},
        {12, 2, 11, 9},
        {18, 8, 14, 4}
    };
    int B[4][4] = {
        {2, 8, 1, 6},
        {7, 4, 9, 0},
        {6, 2, 3, 7},
        {9, 5, 8, 3}
    };

    int h = 4, w = 4;
    int *d_A, *d_B, *d_C, C[h][w];

    hipMalloc((void **)&d_A, h*w*sizeof(int));
    hipMalloc((void **)&d_B, w*w*sizeof(int));
    hipMalloc((void **)&d_C, h*w*sizeof(int));

    hipMemcpy(d_A, A, h*w*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, w*w*sizeof(int), hipMemcpyHostToDevice);

    // a:
    // dim3 threads_a(32, 1, 1);
    // dim3 blocks_a(ceil(h / 32.0), 1, 1);
    // matadd_row<<<blocks_a, threads_a>>>(d_A, d_B, d_C, h, w);

    // b:
    // dim3 threads_b(32, 1, 1);
    // dim3 blocks_b(ceil(w / 32.0), 1, 1);
    // matadd_col<<<blocks_b, threads_b>>>(d_A, d_B, d_C, h, w);

    // c:
    dim3 threads_c (32, 32, 1);
    dim3 blocks_c (ceil(h/32.0), ceil(w/32.0), 1);
    matadd_elem<<<blocks_c, threads_c>>>(d_A, d_B, d_C, h, w);


    hipMemcpy(C, d_C, h*w*sizeof(int), hipMemcpyDeviceToHost);
    printf("Result: ");
    for(int i = 0; i < h; i++) {
        for(int j = 0; j < w; j++) {
            printf("%d  ", C[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}


// Result: 7  23  3  20  
// 16  17  22  5  
// 18  4  14  16  
// 27  13  22  7 