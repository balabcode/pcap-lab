#include <hip/hip_runtime.h>

#include <stdio.h>

#define N 64

// Write a program in CUDA to add two vectors of length N using
// a) block size as N
// b) N threads

__global__ void add(int *a, int *b, int *c, int n)
{
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}

int main(void) {
    int a[N], b[N], c[N], size=sizeof(int);
    int *d_a, *d_b, *d_c;

    hipMalloc((void **) &d_a, N * size);
    hipMalloc((void **) &d_b, N * size);
    hipMalloc((void **) &d_c, N * size);

    for(int i = 0; i < N; i++) a[i] = i*2;
    for(int i = 0; i < N; i++) b[i] = i*3 - 1;

    hipMemcpy(d_a, a, N*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*size, hipMemcpyHostToDevice);

    dim3 blk(1, 1, 1);
    dim3 thr(N, 1, 1);
    add<<<blk, thr>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, N*size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) printf("%d  ", c[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}