// Write a program in CUDA to process a 1D array containing angles in radians to generate
// sine of the angles in the output array. Use appropriate function.


#include <hip/hip_runtime.h>

#include <stdio.h>

#include <math.h>
#define N 16

__global__ void sine(float *a, float *b, int n) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        b[index] = sinf(a[index]);
}

int main(void) {
    float a[N], b[N], size=sizeof(float);
    float *d_a, *d_b;

    hipMalloc((void **) &d_a, N * size);
    hipMalloc((void **) &d_b, N * size);

    for(int i = 0; i < N; i++) a[i] = i*M_PI/N+53.2;
    hipMemcpy(d_a, a, N*size, hipMemcpyHostToDevice);

    dim3 blk(2, 1, 1);
    dim3 thr(8, 1, 1);
    sine<<<blk, thr>>>(d_a, d_b, N);
    hipMemcpy(b, d_b, N*size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) printf("%.2f  ", b[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}

// 0.21  0.01  -0.18  -0.37  -0.55  -0.70  -0.83  -0.92  -0.98  -1.00  -0.98  -0.93  -0.84  -0.71  -0.56  -0.39 