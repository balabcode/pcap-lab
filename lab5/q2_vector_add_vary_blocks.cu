// Implement a CUDA program to add two vectors of length N by keeping the number of
// threads per block as 256 (constant) and vary the number of blocks to handle N elements.

#include <hip/hip_runtime.h>

#include <stdio.h>

#define N 512

__global__ void add(int *a, int *b, int *c, int n)
{
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}

int main(void) {
    int a[N], b[N], c[N], size=sizeof(int);
    int *d_a, *d_b, *d_c;

    hipMalloc((void **) &d_a, N * size);
    hipMalloc((void **) &d_b, N * size);
    hipMalloc((void **) &d_c, N * size);

    for(int i = 0; i < N; i++) a[i] = i*2;
    for(int i = 0; i < N; i++) b[i] = i*3 - 1;

    hipMemcpy(d_a, a, N*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*size, hipMemcpyHostToDevice);

    dim3 blk(ceil(N/256.0), 1, 1);
    dim3 thr(256, 1, 1);
    add<<<blk, thr>>>(d_a, d_b, d_c, N);
    hipMemcpy(c, d_c, N*size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) printf("%d  ", c[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}