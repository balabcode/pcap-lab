#include "hip/hip_runtime.h"
// Write a program in CUDA which performs convolution operation on one-dimensional input
// array N of size width using a mask array M of size mask_width to produce the resultant one-
// dimensional array P of size width.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void convolution_1d(float *N, float *M, float *P, int mask_width, int width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width) {
        float Pval = 0;
        int N_start = i - (mask_width/2);
        for(int j = 0; j < mask_width; j++) {
            if(N_start + j >= 0 && N_start + j < width)
                Pval += N[N_start + j] * M[j];
        }
        P[i] = Pval;
    }
}

int main() {
    int width, mask_width=5;
    float M[] = {3, 4, 5, 4, 3};

    printf("Enter size of array: ");
    scanf("%d", &width);
    float N[width]={0}, P[width]={0};

    printf("Enter elements: ");
    for(int i = 0; i < width; i++) {
        scanf("%f", &N[i]);
    }

    float *d_N, *d_M, *d_P;
    int size = width*sizeof(float);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);
    hipMalloc((void**)&d_M, mask_width*sizeof(float));

    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width*sizeof(float), hipMemcpyHostToDevice);

    dim3 blk(ceil(width/3.0), 1, 1);
    dim3 thr(3, 1, 1);

    convolution_1d<<<blk, thr>>>(d_N, d_M, d_P, mask_width, width);
    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

    printf("Output from 1D Convolution:\n");
    for(int i = 0; i < width; i++) {
        printf("%.2f  ", P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}


// student@dbl-35:~/Documents/220962448/lab6$ ./out/q1
// Enter size of array: 7
// Enter elements: 1 2 3 4 5 6 7
// Output from 1D Convolution:
// 22.00  38.00  57.00  76.00  95.00  90.00  74.00