#include "hip/hip_runtime.h"
// Write a program in CUDA to perform selection sort in parallel

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>


__global__ void selection_sort(int *N, int *P, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int count = 0;
        for(int j = 0; j < n; j++) {
            if(N[j] < N[i]) count++;
        }
        P[i] = count;
    }
}

__global__ void place_elements(int *N, int *P, int *sorted, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n) {
        sorted[P[i]] = N[i];
    }
}

int main() {
    int n;

    printf("Enter number of elems: ");
    scanf("%d", &n);

    int N[n]={0}, P[n]={0}, size=n*sizeof(int);
    printf("Enter elems:\n");
    for(int i = 0; i < n; i++) scanf("%d", &N[i]);

    int *d_N, *d_P, *d_sorted;
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);
    hipMalloc((void**)&d_sorted, size);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, size, hipMemcpyHostToDevice);

    dim3 blk(ceil(n/3.0), 1, 1);
    dim3 thr(3, 1, 1);
    selection_sort<<<blk, thr>>>(d_N, d_P, n);
    place_elements<<<blk, thr>>>(d_N, d_P, d_sorted, n);

    hipMemcpy(P, d_sorted, size, hipMemcpyDeviceToHost);
    
    printf("Sorted Array:\n");
    for(int i = 0; i < n; i++) {
        printf("%d ", P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_P);
    hipFree(d_sorted);
    return 0;
}

// student@dbl-35:~/Documents/220962448/lab6$ ./out/q2
// Enter number of elems: 12
// Enter elems:
// 5 7 2 3 4 1 8 9 6 11 10 12
// Sorted Array:
// 1 2 3 4 5 6 7 8 9 10 11 12 
