#include "hip/hip_runtime.h"
// Write a program in CUDA to perform odd even transposition sort in parallel

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void odd_sort(int *N, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n-1 && i % 2 != 0) {
        if(N[i] > N[i+1]) {
            int temp = N[i];
            N[i] = N[i+1];
            N[i+1] = temp;
        }
    }
}
__global__ void even_sort(int *N, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n-1 && i % 2 == 0) {
        if(N[i] > N[i+1]) {
            int temp = N[i];
            N[i] = N[i+1];
            N[i+1] = temp;
        }
    }
}

int main() {
    int n;

    printf("Enter number of elems: ");
    scanf("%d", &n);

    int N[n]={0}, size=n*sizeof(int);
    printf("Enter elems:\n");
    for(int i = 0; i < n; i++) scanf("%d", &N[i]);

    int *d_N;
    hipMalloc((void**)&d_N, size);
    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);

    dim3 blk(ceil(n/3.0), 1, 1);
    dim3 thr(3, 1, 1);

    for(int i = 0; i < n/2; i++) {
        odd_sort<<<blk, thr>>>(d_N, n);
        even_sort<<<blk, thr>>>(d_N, n);
    }

    hipMemcpy(N, d_N, size, hipMemcpyDeviceToHost);

    printf("Sorted Array:\n");
    for(int i = 0; i < n; i++) printf("%d ", N[i]);
    printf("\n");

    hipFree(d_N);
    return 0;
}

// student@dbl-35:~/Documents/220962448/lab6$ ./out/q3
// Enter number of elems: 12
// Enter elems:
// 3 5 6 2 7 4 1 0 8 10 9 11
// Sorted Array:
// 0 1 2 3 4 5 6 7 8 9 10 11 