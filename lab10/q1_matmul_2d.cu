#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void matmul_2d(int *A, int *B, int *C, int hA, int wA, int wB) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < hA && col < wB) {
        int elem = 0;
        for(int k = 0; k < wA; k++) {
            elem += A[row*wA + k] * B[k*wB + col];
        }
        C[row*wB + col] = elem;
    }
}

int main() {
    int A[4][4] = {
        {3, 7, 1, 8},
        {2, 9, 4, 5},
        {6, 0, 8, 2},
        {9, 3, 6, 1}
    };
    int B[4][4] = {
        {2, 8, 1, 6},
        {7, 4, 9, 0},
        {6, 2, 3, 7},
        {9, 5, 8, 3}
    };

    int hA = 4, wA = 4, wB = 4;
    int *d_A, *d_B, *d_C, C[hA][wB];

    hipMalloc((void **)&d_A, hA*wA*sizeof(int));
    hipMalloc((void **)&d_B, wA*wB*sizeof(int));
    hipMalloc((void **)&d_C, hA*wB*sizeof(int));

    hipMemcpy(d_A, A, hA*wA*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, wA*wB*sizeof(int), hipMemcpyHostToDevice);

    dim3 threads_c (32, 32, 1);
    dim3 blocks_c (ceil(hA/32.0), ceil(wB/32.0), 1);
    matmul_2d<<<blocks_c, threads_c>>>(d_A, d_B, d_C, hA, wA, wB);


    hipMemcpy(C, d_C, hA*wB*sizeof(int), hipMemcpyDeviceToHost);
    printf("Result: ");
    for(int i = 0; i < hA; i++) {
        for(int j = 0; j < wB; j++) {
            printf("%d  ", C[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

// Result: 133  94  133  49  
// 136  85  135  55  
// 78  74  46  98  
// 84  101  62  99 