#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>

__constant__ float d_M_constant[5]; //NEW

__global__ void convolution_1d(float *N, float *P, int mask_width, int width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width) {
        float Pval = 0;
        int N_start = i - (mask_width / 2);
        for (int j = 0; j < mask_width; j++) {
            if (N_start + j >= 0 && N_start + j < width)
                Pval += N[N_start + j] * d_M_constant[j];  //NEW
        }
        P[i] = Pval;
    }
}

int main() {
    int width, mask_width = 5;
    float M[] = {3, 4, 5, 4, 3};

    printf("Enter size of array: ");
    scanf("%d", &width);
    float N[width] = {0}, P[width] = {0};

    printf("Enter elements: ");
    for (int i = 0; i < width; i++) {
        scanf("%f", &N[i]);
    }

    float *d_N, *d_P;
    int size = width * sizeof(float);
    hipMalloc((void **)&d_N, size);
    hipMalloc((void **)&d_P, size);

    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, size, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(d_M_constant), M, mask_width * sizeof(float)); //NEW

    dim3 blk(ceil((float)width / 3.0), 1, 1);
    dim3 thr(3, 1, 1);

    convolution_1d<<<blk, thr>>>(d_N, d_P, mask_width, width);
    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

    printf("Output from 1D Convolution:\n");
    for (int i = 0; i < width; i++) {
        printf("%.2f  ", P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_P);

    return 0;
}

// student@dbl-35:~/Documents/220962448/lab10$ ./out/q2
// Enter size of array: 7
// Enter elements: 1 2 3 4 5 6 7
// Output from 1D Convolution:
// 22.00  38.00  57.00  76.00  95.00  90.00  74.00