// Write a program in CUDA to perform matrix multiplication using 2D Grid and 2D Block.

#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void matmul_shared(int *A, int *B, int *C, int hA, int wA, int wB) {
    int row = blockIdx.y * blockDim.y + threadIdx.x;
    int col = blockIdx.x * blockDim.x + threadIdx.y;

    if (row < hA && col < wB) {
        int sum = 0;
        __shared__ int sA[16][16];
        __shared__ int sB[16][16];

        for (int k = 0; k < wA; k += blockDim.y) {
            int aRow = row, aCol = k + threadIdx.y;
            int bRow = k + threadIdx.x, bCol = col;

            sA[threadIdx.x][threadIdx.y] = (aRow < hA && aCol < wA) ? A[aRow * wA + aCol] : 0;
            sB[threadIdx.x][threadIdx.y] = (bRow < wA && bCol < wB) ? B[bRow * wB + bCol] : 0;

            __syncthreads();

            for (int j = 0; j < blockDim.y; ++j) {
                sum += sA[threadIdx.x][j] * sB[j][threadIdx.y]; 
            }
            __syncthreads();
        }
        C[row * wB + col] = sum;
    }
}

int main() {
    int A[5][4] = {
        {3, 7, 1, 8},
        {2, 9, 4, 5},
        {6, 0, 8, 2},
        {9, 3, 6, 1},
        {4, 5, 7, 9}
    };
    int B[4][6] = {
        {2, 8, 1, 6, 9, 3},
        {7, 4, 9, 0, 5, 8},
        {6, 2, 3, 7, 1, 4},
        {9, 5, 8, 3, 7, 6}
    };

    int hA = 5, wA = 4, wB = 6;
    int *d_A, *d_B, *d_C, C[hA][wB];

    hipMalloc((void **)&d_A, hA*wA*sizeof(int));
    hipMalloc((void **)&d_B, wA*wB*sizeof(int));
    hipMalloc((void **)&d_C, hA*wB*sizeof(int));

    hipMemcpy(d_A, A, hA*wA*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, wA*wB*sizeof(int), hipMemcpyHostToDevice);

    dim3 threads_c (16, 16, 1);
    dim3 blocks_c (ceil(wB/16.0), ceil(hA/16.0), 1);
    matmul_shared<<<blocks_c, threads_c>>>(d_A, d_B, d_C, hA, wA, wB);


    hipMemcpy(C, d_C, hA*wB*sizeof(int), hipMemcpyDeviceToHost);
    printf("Result: \n");
    for(int i = 0; i < hA; i++) {
        for(int j = 0; j < wB; j++) {
            printf("%d  ", C[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

// student@dbl-35:~/Documents/220962448/lab10$ ./out/q1
// Result: 
// 133  94  133  49  119  117  
// 136  85  135  55  102  124  
// 78  74  46  98  76  62  
// 84  101  62  99  109  81  
// 166  111  142  100  131  134  
