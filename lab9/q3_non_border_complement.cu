// Write a CUDA program that reads a matrix A of size MXN and produce an output matrix B of
// same size such that it replaces all the non-border elements (numbers in bold) of A with its equivalent
// 1’s complement and remaining elements same as matrix A.

#include <stdio.h>
#include <hip/hip_runtime.h>



__device__ int ones_complement(int n) {
    int bin=0;
    int inc=1;
    for(int i = n; i > 0; i /= 2) {
        bin += (i%2 == 0)*inc;
        inc *= 10;
    }
    return bin;
}

__global__ void non_border_complement(int *mat, int *out, int rows, int cols) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;

    if(r < rows && c < cols) {
        int elem;
        if (r == 0 || r == rows-1 || c == 0 || c == cols-1) {
            elem = mat[r*cols + c];
        } else {
            elem = ones_complement(mat[r*cols + c]);
        }
        out[r*cols + c] = elem;
    }
}

int main() {
    int r, c;
    printf("Rows: ");
    scanf("%d", &r);
    printf("Columns: ");
    scanf("%d", &c);

    int mat[r][c], out[r][c];

    printf("Enter matrix:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            scanf("%d", &mat[i][j]);
        }
    }

    int *d_mat, *d_out;
    hipMalloc((void **)&d_mat, r * c * sizeof(int));
    hipMalloc((void **)&d_out, r * c * sizeof(int));

    hipMemcpy(d_mat, mat, r * c * sizeof(int), hipMemcpyHostToDevice);

    non_border_complement<<<dim3(ceil(r/32.0), ceil(c/32.0)), dim3(32, 32)>>>(d_mat, d_out, r, c);
    hipMemcpy(out, d_out, r * c * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            printf("%d ", out[i][j]);
        }
        printf("\n");
    }

    hipFree(d_mat);
    hipFree(d_out);
    return 0;
}

// student@dbl-35:~/Documents/220962448/lab9$ ./out/q3
// Rows: 4               
// Columns: 4
// Enter matrix:
// 1 2 3 4 6 5 8 3 2 4 10 1 9 1 2 5
// Result:
// 1 2 3 4 
// 6 10 111 3 
// 2 11 101 1 
// 9 1 2 5 