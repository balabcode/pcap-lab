// Write a program in CUDA to read MXN matrix A and replace 1“ row of this matrix by same
// elements, 2"¢ row elements by square of each element and 3" row elements by cube of each element
// and so on.

#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void matrix_format(int *mat, int *out, int r, int c) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < r) {
        int elem;
        for(int i = 0; i < c; i++) {
            elem = (int)pow((float)mat[row*c + i], (float)row+1);
            out[row*c + i] = elem;
        }
    }
}


int main() {
    int r, c;
    printf("Rows: ");
    scanf("%d", &r);
    printf("Columns: ");
    scanf("%d", &c);

    int mat[r][c], out[r][c];

    printf("Enter matrix:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            scanf("%d", &mat[i][j]);
        }
    }

    int *d_mat, *d_out;
    hipMalloc((void **)&d_mat, r * c * sizeof(int));
    hipMalloc((void **)&d_out, r * c * sizeof(int));

    hipMemcpy(d_mat, mat, r * c * sizeof(int), hipMemcpyHostToDevice);

    matrix_format<<<dim3(ceil(r/32.0)), dim3(32)>>>(d_mat, d_out, r, c);
    hipMemcpy(out, d_out, r * c * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            printf("%d ", out[i][j]);
        }
        printf("\n");
    }

    hipFree(d_mat);
    hipFree(d_out);
    return 0;
}

// student@dbl-35:~/Documents/220962448/lab9$ ./out/q2
// Rows: 3
// Columns: 3
// Enter matrix:
// 1 2 3 1 2 3 1 2 3
// Result:
// 1 2 3 
// 1 4 9 
// 1 8 27