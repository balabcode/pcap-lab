// Write a program in CUDA to perform parallel Sparse Matrix - Vector multiplication using com-
// pressed sparse row (CSR) storage format. Represent the input sparse matrix in CSR format in the
// host code.

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>

__global__ void spvm(int *values, int *col_indices, int *row_ptrs, int *x, int *y, int num_rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        int start = row_ptrs[row];
        int end = row_ptrs[row + 1];
        int sum = 0;
        for (int j = start; j < end; j++) {
            sum += values[j] * x[col_indices[j]];
        }
        y[row] = sum;
    }
}

int main() {
    int r, c;
    printf("Rows: ");
    scanf("%d", &r);
    printf("Columns: ");
    scanf("%d", &c);

    int mat[r][c];

    printf("Enter matrix:\n");
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            scanf("%d", &mat[i][j]);
        }
    }

    int vec[c];
    printf("Enter vector of size %d:\n", c);
    for (int i = 0; i < c; i++) {
        scanf("%d", &vec[i]);
    }

    int non_zero = 0;
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            if (mat[i][j] != 0) {
                non_zero++;
            }
        }
    }

    int *h_values = (int *)malloc(non_zero * sizeof(int));
    int *h_col_indices = (int *)malloc(non_zero * sizeof(int));
    int *h_row_ptrs = (int *)malloc((r + 1) * sizeof(int));

    int idx = 0;
    for (int i = 0; i < r; i++) {
        h_row_ptrs[i] = idx;
        for (int j = 0; j < c; j++) {
            if (mat[i][j] != 0) {
                h_values[idx] = mat[i][j];
                h_col_indices[idx] = j;
                idx++;
            }
        }
    }
    h_row_ptrs[r] = non_zero;

    int *d_values, *d_col_indices, *d_row_ptrs, *d_x, *d_y;
    hipMalloc((void**)&d_values, non_zero * sizeof(int));
    hipMalloc((void**)&d_col_indices, non_zero * sizeof(int));
    hipMalloc((void**)&d_row_ptrs, (r + 1) * sizeof(int));
    hipMalloc((void**)&d_x, c * sizeof(int));
    hipMalloc((void**)&d_y, r * sizeof(int));

    hipMemcpy(d_values, h_values, non_zero * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, h_col_indices, non_zero * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptrs, h_row_ptrs, (r + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, vec, c * sizeof(int), hipMemcpyHostToDevice);

    spvm<<<dim3(ceil(r/32.0)), dim3(32)>>>(d_values, d_col_indices, d_row_ptrs, d_x, d_y, r);

    int *h_y = (int *)malloc(r * sizeof(int));
    hipMemcpy(h_y, d_y, r * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < r; i++) {
        printf("%d ", h_y[i]);
    }
    printf("\n");

    free(h_values);
    free(h_col_indices);
    free(h_row_ptrs);
    free(h_y);
    hipFree(d_values);
    hipFree(d_col_indices);
    hipFree(d_row_ptrs);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}

// student@dbl-35:~/Documents/220962448/lab9$ ./out/q1
// Rows: 3
// Columns: 3
// Enter matrix:
// 0 0 1 2 0 0 0 3 0
// Enter vector of size 3:
// 2 0 4
// Result:
// 4 4 0 