#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>

__constant__ float d_M_constant[5];

__global__ void convolution_1d(float *N, float *P, int mask_width, int width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width) {
        int mask_radius = mask_width / 2;
        int tile_size = blockDim.x + 2 * mask_radius;
        __shared__ float s_N[7];
        
        int block_start = blockIdx.x * blockDim.x - mask_radius;
        
        for (int j = 0; j < (tile_size + blockDim.x - 1) / blockDim.x; j++) {
            int load_index = threadIdx.x + j * blockDim.x;
            if (load_index < tile_size) {
                int global_index = block_start + load_index;
                s_N[load_index] = (global_index >= 0 && global_index < width) ? N[global_index] : 0.0f;
            }
        }
        __syncthreads();

        float Pval = 0.0f;
        for (int j = 0; j < mask_width; j++) {
            Pval += s_N[threadIdx.x + j] * d_M_constant[j];
        }
        P[i] = Pval;
    }
}

int main() {
    int width, mask_width = 5;
    float M[] = {3, 4, 5, 4, 3};

    printf("Enter size of array: ");
    scanf("%d", &width);
    float N[width], P[width];

    printf("Enter elements: ");
    for (int i = 0; i < width; i++) {
        scanf("%f", &N[i]);
    }

    float *d_N, *d_P;
    int size = width * sizeof(float);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);

    hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_M_constant), M, mask_width * sizeof(float));

    dim3 blk(ceil((float)width / 3.0), 1, 1);
    dim3 thr(3, 1, 1);

    convolution_1d<<<blk, thr>>>(d_N, d_P, mask_width, width);
    hipMemcpy(P, d_P, size, hipMemcpyDeviceToHost);

    printf("Output from 1D Convolution:\n");
    for (int i = 0; i < width; i++) {
        printf("%.2f ", P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_P);
    return 0;
}

// student@dbl-35:~/Documents/220962448/lab10$ ./out/q2
// Enter size of array: 7
// Enter elements: 1 2 3 4 5 6 7
// Output from 1D Convolution:
// 22.00  38.00  57.00  76.00  95.00  90.00  74.00