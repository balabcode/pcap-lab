#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void inclusive_scan(int *arr, int *out, int n) {
    extern __shared__ int s_data[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    
    s_data[tid] = (i < n) ? arr[i] : 0;
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int val = (tid >= stride) ? s_data[tid - stride] : 0;
        __syncthreads();
        s_data[tid] += val;
        __syncthreads();
    }

    if (i < n) out[i] = s_data[tid];
}

int main() {
    int n = 8;
    int arr[] = {1, 2, 3, 4, 5, 6, 7, 8};
    int out[8] = {0};
    int *d_arr, *d_out;

    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMalloc((void**)&d_out, n * sizeof(int));

    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blocks(ceil(n / 16.0), 1, 1);
    dim3 threads(16, 1, 1);
    inclusive_scan<<<blocks, threads, threads.x * sizeof(int)>>>(d_arr, d_out, n);

    hipMemcpy(out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Input array: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    printf("Inclusive scan result: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", out[i]);
    }
    printf("\n");

    hipFree(d_arr);
    hipFree(d_out);

    return 0;
}
